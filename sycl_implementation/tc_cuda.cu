#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <chrono>
#include <math.h>
#include <iomanip>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/unique.h>
#include <thrust/copy.h>
#include <thrust/fill.h>

using namespace std;
#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

struct Entity {
    int key;
    int value;
};

struct Output {
    int block_size;
    int grid_size;
    long int input_rows;
    long int hashtable_rows;
    double load_factor;
    double initialization_time;
    double memory_clear_time;
    double read_time;
    double reverse_time;
    double hashtable_build_time;
    long int hashtable_build_rate;
    double join_time;
    double projection_time;
    double deduplication_time;
    double union_time;
    double total_time;
    const char *dataset_name;
} output;

struct KernelTimer {
    hipEvent_t start;
    hipEvent_t stop;

    KernelTimer() {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~KernelTimer() {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void start_timer() {
        hipEventRecord(start, 0);
    }

    void stop_timer() {
        hipEventRecord(stop, 0);
    }

    float get_spent_time() {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        elapsed /= 1000.0;
        return elapsed;
    }
};

struct is_equal {
    __host__ __device__
    bool operator()(const Entity &lhs, const Entity &rhs) {
        if ((lhs.key == rhs.key) && (lhs.value == rhs.value))
            return true;
        return false;
    }
};


struct cmp {
    __host__ __device__
    bool operator()(const Entity &lhs, const Entity &rhs) {
        if (lhs.key < rhs.key)
            return true;
        else if (lhs.key > rhs.key)
            return false;
        else {
            if (lhs.value < rhs.value)
                return true;
            else if (lhs.value > rhs.value)
                return false;
            return true;
        }
    }
};

__device__ int get_position(int key, int hash_table_row_size) {
    key ^= key >> 16;
    key *= 0x85ebca6b;
    key ^= key >> 13;
    key *= 0xc2b2ae35;
    key ^= key >> 16;
    return key & (hash_table_row_size - 1);
}

void show_time_spent(string message,
                     chrono::high_resolution_clock::time_point time_point_begin,
                     chrono::high_resolution_clock::time_point time_point_end) {
    chrono::duration<double> time_span = time_point_end - time_point_begin;
    cout << message << ": " << time_span.count() << " seconds" << endl;
}

double get_time_spent(string message,
                      chrono::high_resolution_clock::time_point time_point_begin,
                      chrono::high_resolution_clock::time_point time_point_end) {
    chrono::duration<double> time_span = time_point_end - time_point_begin;
    if (message != "")
        cout << message << ": " << time_span.count() << " seconds" << endl;
    return time_span.count();
}

void show_relation(int *data, int total_rows,
                   int total_columns, const char *relation_name,
                   int visible_rows, int skip_zero) {
    int count = 0;
    cout << "Relation name: " << relation_name << endl;
    cout << "===================================" << endl;
    for (int i = 0; i < total_rows; i++) {
        int skip = 0;
        for (int j = 0; j < total_columns; j++) {
            if ((skip_zero == 1) && (data[(i * total_columns) + j] == 0)) {
                skip = 1;
                continue;
            }
            cout << data[(i * total_columns) + j] << " ";
        }
        if (skip == 1)
            continue;
        cout << endl;
        count++;
        if (count == visible_rows) {
            cout << "Result cropped at row " << count << "\n" << endl;
            return;
        }

    }
    cout << "Result counts " << count << "\n" << endl;
    cout << "" << endl;
}

int *get_relation_from_file(const char *file_path, int total_rows, int total_columns, char separator) {
    int *data = (int *) malloc(total_rows * total_columns * sizeof(int));
    FILE *data_file = fopen(file_path, "r");
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            if (j != (total_columns - 1)) {
                fscanf(data_file, "%d%c", &data[(i * total_columns) + j], &separator);
            } else {
                fscanf(data_file, "%d", &data[(i * total_columns) + j]);
            }
        }
    }
    return data;
}

void get_relation_from_file_gpu(int *data, const char *file_path, int total_rows, int total_columns, char separator) {
    FILE *data_file = fopen(file_path, "r");
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            if (j != (total_columns - 1)) {
                fscanf(data_file, "%d%c", &data[(i * total_columns) + j], &separator);
            } else {
                fscanf(data_file, "%d", &data[(i * total_columns) + j]);
            }
        }
    }
}


void get_random_relation(int *data, int total_rows, int total_columns) {
    for (int i = 0; i < total_rows; i++) {
        for (int j = 0; j < total_columns; j++) {
            data[(i * total_columns) + j] = (rand() % (32767 - 0 + 1)) + 0;
        }
    }
}

void get_string_relation(int *data, int total_rows, int total_columns) {
    int x = 1, y = 2;
    for (int i = 0; i < total_rows; i++) {
        data[(i * total_columns) + 0] = x++;
        data[(i * total_columns) + 1] = y++;
    }
}

void get_reverse_relation_gpu(int *reverse_data, int *data, int total_rows, int total_columns) {
    for (int i = 0; i < total_rows; i++) {
        int pos = total_columns - 1;
        for (int j = 0; j < total_columns; j++) {
            reverse_data[(i * total_columns) + j] = data[(i * total_columns) + pos];
            pos--;
        }
    }
}


void show_hash_table(Entity *hash_table, long int hash_table_row_size, const char *hash_table_name) {
    int count = 0;
    cout << "Hashtable name: " << hash_table_name << endl;
    cout << "===================================" << endl;
    for (int i = 0; i < hash_table_row_size; i++) {
        if (hash_table[i].key != -1) {
            cout << hash_table[i].key << " " << hash_table[i].value << endl;
            count++;
        }
    }
    cout << "Row counts " << count << "\n" << endl;
    cout << "" << endl;
}

void show_entity_array(Entity *data, int data_rows, const char *array_name) {
    long int count = 0;
    cout << "Entity name: " << array_name << endl;
    cout << "===================================" << endl;
    for (int i = 0; i < data_rows; i++) {
        if (data[i].key != -1) {
            cout << data[i].key << " " << data[i].value << endl;
            count++;
        }
    }
    cout << "Row counts " << count << "\n" << endl;
    cout << "" << endl;
}

long int get_row_size(const char *data_path) {
    long int row_size = 0;
    int base = 1;
    for (int i = strlen(data_path) - 1; i >= 0; i--) {
        if (isdigit(data_path[i])) {
            int digit = (int) data_path[i] - '0';
            row_size += base * digit;
            base *= 10;
        }
    }
    return row_size;
}

/*
 * Method that returns position in the hashtable for a key using Murmur3 hash
 * */


__global__
void build_hash_table(Entity *hash_table, long int hash_table_row_size,
                      int *relation, long int relation_rows, int relation_columns) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_rows; i += stride) {
        int key = relation[(i * relation_columns) + 0];
        int value = relation[(i * relation_columns) + 1];
        int position = get_position(key, hash_table_row_size);
        while (true) {
            int existing_key = atomicCAS(&hash_table[position].key, -1, key);
            if (existing_key == -1) {
                hash_table[position].value = value;
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
    }
}

__global__
void initialize_result_t_delta(Entity *result, Entity *t_delta,
                               int *relation, long int relation_rows, int relation_columns) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_rows; i += stride) {
        t_delta[i].key = result[i].key = relation[(i * relation_columns) + 0];
        t_delta[i].value = result[i].value = relation[(i * relation_columns) + 1];
    }
}

__global__
void copy_struct(Entity *source, long int source_rows, Entity *destination) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= source_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < source_rows; i += stride) {
        destination[i].key = source[i].key;
        destination[i].value = source[i].value;
    }
}

__global__
void negative_fill_struct(Entity *source, long int source_rows) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= source_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < source_rows; i += stride) {
        source[i].key = -1;
        source[i].value = -1;
    }
}

__global__
void get_reverse_relation(int *relation, long int relation_rows, int relation_columns, Entity *t_delta) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (long int i = index; i < relation_rows; i += stride) {
        t_delta[i].key = relation[(i * relation_columns) + 0];
        t_delta[i].value = relation[(i * relation_columns) + 1];
    }
}


__global__
void get_join_result_size(Entity *hash_table, long int hash_table_row_size,
                          Entity *t_delta, long int relation_rows,
                          int *join_result_size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_rows; i += stride) {
        int key = t_delta[i].value;
        int current_size = 0;
        int position = get_position(key, hash_table_row_size);
        while (true) {
            if (hash_table[position].key == key) {
                current_size++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
        join_result_size[i] = current_size;
    }
}

__global__
void get_join_result(Entity *hash_table, int hash_table_row_size,
                     Entity *t_delta, int relation_rows, int *offset, Entity *join_result) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < relation_rows; i += stride) {
        int key = t_delta[i].value;
        int value = t_delta[i].key;
        int start_index = offset[i];
        int position = get_position(key, hash_table_row_size);
        while (true) {
            if (hash_table[position].key == key) {
                join_result[start_index].key = value;
                join_result[start_index].value = hash_table[position].value;
                start_index++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
    }
}

__global__
void get_join_result_size_ar(Entity *hash_table, long int hash_table_row_size,
                             int *t_delta, long int relation_rows,
                             int *join_result_size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;

    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < relation_rows; i += stride) {
        int key = t_delta[(i * 2) + 1];
        int current_size = 0;
        int position = get_position(key, hash_table_row_size);
        while (true) {
            if (hash_table[position].key == key) {
                current_size++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
        join_result_size[i] = current_size;
    }
}

__global__
void get_join_result_ar(Entity *hash_table, int hash_table_row_size,
                        int *t_delta, int relation_rows, int *offset, Entity *join_result) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= relation_rows) return;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < relation_rows; i += stride) {
        int key = t_delta[(i * 2) + 1];
        int value = t_delta[i * 2];
        int start_index = offset[i];
        int position = get_position(key, hash_table_row_size);
        while (true) {
            if (hash_table[position].key == key) {
                join_result[start_index].key = value;
                join_result[start_index].value = hash_table[position].value;
                start_index++;
            } else if (hash_table[position].key == -1) {
                break;
            }
            position = (position + 1) & (hash_table_row_size - 1);
        }
    }
}


void gpu_tc(const char *data_path, char separator,
            long int relation_rows, double load_factor,
            int preferred_grid_size, int preferred_block_size, const char *dataset_name, int number_of_sm) {
    int relation_columns = 2;
    std::chrono::high_resolution_clock::time_point time_point_begin;
    std::chrono::high_resolution_clock::time_point time_point_end;
    std::chrono::high_resolution_clock::time_point temp_time_begin;
    std::chrono::high_resolution_clock::time_point temp_time_end;
    KernelTimer timer;
    time_point_begin = chrono::high_resolution_clock::now();
    double spent_time;
    output.initialization_time = 0;
    output.join_time = 0;
    output.projection_time = 0;
    output.deduplication_time = 0;
    output.memory_clear_time = 0;
    output.union_time = 0;
    output.total_time = 0;
    double sort_time = 0.0;
    double unique_time = 0.0;
    double merge_time = 0.0;
    double temp_spent_time = 0.0;

    int block_size, grid_size;
    int *relation;
    int *relation_host;
    Entity *hash_table, *result, *t_delta;
    Entity *result_host;
    long int join_result_rows;
    long int t_delta_rows = relation_rows;
    long int result_rows = relation_rows;
    long int iterations = 0;
    long int hash_table_rows = (long int) relation_rows / load_factor;
    hash_table_rows = pow(2, ceil(log(hash_table_rows) / log(2)));

    checkCuda(hipHostMalloc((void **) &relation_host, relation_rows * relation_columns * sizeof(int)));
    checkCuda(hipMalloc((void **) &relation, relation_rows * relation_columns * sizeof(int)));
    checkCuda(hipMalloc((void **) &result, result_rows * sizeof(Entity)));
    checkCuda(hipMalloc((void **) &t_delta, relation_rows * sizeof(Entity)));
    checkCuda(hipMalloc((void **) &hash_table, hash_table_rows * sizeof(Entity)));

    // Block size is 512 if preferred_block_size is 0
    block_size = 512;
    // Grid size is 32 times of the number of streaming multiprocessors if preferred_grid_size is 0
    grid_size = 32 * number_of_sm;
    if (preferred_grid_size != 0) {
        grid_size = preferred_grid_size;
    }
    if (preferred_block_size != 0) {
        block_size = preferred_block_size;
    }
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.initialization_time += spent_time;
    time_point_begin = chrono::high_resolution_clock::now();
    get_relation_from_file_gpu(relation_host, data_path,
                               relation_rows, relation_columns, separator);
    hipMemcpy(relation, relation_host, relation_rows * relation_columns * sizeof(int),
               hipMemcpyHostToDevice);
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.read_time = spent_time;

    Entity negative_entity;
    negative_entity.key = -1;
    negative_entity.value = -1;
    time_point_begin = chrono::high_resolution_clock::now();
    thrust::fill(thrust::device, hash_table, hash_table + hash_table_rows, negative_entity);
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.initialization_time += spent_time;
    timer.start_timer();
    build_hash_table<<<grid_size, block_size>>>
            (hash_table, hash_table_rows,
             relation, relation_rows,
             relation_columns);
    checkCuda(hipDeviceSynchronize());
    timer.stop_timer();
    spent_time = timer.get_spent_time();
    output.hashtable_build_time = spent_time;
    output.hashtable_build_rate = (double) relation_rows / spent_time;
    output.join_time += spent_time;

    timer.start_timer();
    // initial result and t delta both are same as the input relation
    initialize_result_t_delta<<<grid_size, block_size>>>(result, t_delta, relation, relation_rows, relation_columns);
    checkCuda(hipDeviceSynchronize());
    timer.stop_timer();
    spent_time = timer.get_spent_time();
    output.union_time += spent_time;
    temp_time_begin = chrono::high_resolution_clock::now();
    thrust::stable_sort(thrust::device, result, result + relation_rows, cmp());
    temp_time_end = chrono::high_resolution_clock::now();
    temp_spent_time = get_time_spent("", temp_time_begin, temp_time_end);
    sort_time += temp_spent_time;
    output.deduplication_time += temp_spent_time;

    time_point_begin = chrono::high_resolution_clock::now();
    hipFree(relation);
    hipHostFree(relation_host);
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.memory_clear_time += spent_time;

    // Run the fixed point iterations for transitive closure computation
    while (true) {
        double temp_join = 0.0, temp_union = 0.0, temp_deduplication = 0.0, temp_memory_clear = 0.0;
        double temp_merge = 0.0, temp_sort = 0.0, temp_unique = 0.0;
        time_point_begin = chrono::high_resolution_clock::now();
        int *offset;
        Entity *join_result;
        checkCuda(hipMalloc((void **) &offset, t_delta_rows * sizeof(int)));
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_join += spent_time;
        output.join_time += spent_time;
        timer.start_timer();
        // First pass to get the join result size for each row of t_delta
        get_join_result_size<<<grid_size, block_size>>>(hash_table, hash_table_rows, t_delta, t_delta_rows,
                                                        offset);
        checkCuda(hipDeviceSynchronize());
        timer.stop_timer();
        spent_time = timer.get_spent_time();
        temp_join += spent_time;
        output.join_time += spent_time;
        time_point_begin = chrono::high_resolution_clock::now();
        join_result_rows = thrust::reduce(thrust::device, offset, offset + t_delta_rows, 0);
        thrust::exclusive_scan(thrust::device, offset, offset + t_delta_rows, offset);
        checkCuda(hipMalloc((void **) &join_result, join_result_rows * sizeof(Entity)));
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_join += spent_time;
        output.join_time += spent_time;
        timer.start_timer();
        // Second pass to generate the join result of t_delta and the hash_table
        get_join_result<<<grid_size, block_size>>>(hash_table, hash_table_rows,
                                                   t_delta, t_delta_rows, offset, join_result);
        checkCuda(hipDeviceSynchronize());
        timer.stop_timer();
        spent_time = timer.get_spent_time();
        temp_join += spent_time;
        output.join_time += spent_time;
        // deduplication of projection
        // first sort the array and then remove consecutive duplicated elements
        temp_time_begin = chrono::high_resolution_clock::now();
        thrust::stable_sort(thrust::device, join_result, join_result + join_result_rows, cmp());
        temp_time_end = chrono::high_resolution_clock::now();
        temp_spent_time = get_time_spent("", temp_time_begin, temp_time_end);
        temp_sort += temp_spent_time;
        temp_deduplication += temp_spent_time;
        sort_time += temp_spent_time;
        output.deduplication_time += temp_spent_time;
        temp_time_begin = chrono::high_resolution_clock::now();
        long int projection_rows = (thrust::unique(thrust::device,
                                                   join_result, join_result + join_result_rows,
                                                   is_equal())) - join_result;
        temp_time_end = chrono::high_resolution_clock::now();
        temp_spent_time = get_time_spent("", temp_time_begin, temp_time_end);
        temp_unique += temp_spent_time;
        temp_deduplication += temp_spent_time;
        unique_time += temp_spent_time;
        output.deduplication_time += temp_spent_time;
        time_point_begin = chrono::high_resolution_clock::now();
        hipFree(t_delta);
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_memory_clear += spent_time;
        output.memory_clear_time += spent_time;
        time_point_begin = chrono::high_resolution_clock::now();
        checkCuda(hipMalloc((void **) &t_delta, projection_rows * sizeof(Entity)));
        thrust::copy(thrust::device, join_result, join_result + projection_rows, t_delta);
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_join += spent_time;
        output.join_time += spent_time;

        time_point_begin = chrono::high_resolution_clock::now();
        Entity *concatenated_result;
        long int concatenated_rows = projection_rows + result_rows;
        checkCuda(hipMalloc((void **) &concatenated_result, concatenated_rows * sizeof(Entity)));
        temp_time_begin = chrono::high_resolution_clock::now();
        // merge two sorted array: previous result and join result
        thrust::merge(thrust::device,
                      result, result + result_rows,
                      join_result, join_result + projection_rows,
                      concatenated_result, cmp());
        temp_time_end = chrono::high_resolution_clock::now();
        temp_spent_time = get_time_spent("", temp_time_begin, temp_time_end);
        temp_merge += temp_spent_time;
        merge_time += temp_spent_time;
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_union += spent_time;
        output.union_time += spent_time;
        long int deduplicated_result_rows;
        temp_time_begin = chrono::high_resolution_clock::now();
        deduplicated_result_rows = (thrust::unique(thrust::device,
                                                   concatenated_result,
                                                   concatenated_result + concatenated_rows,
                                                   is_equal())) - concatenated_result;
        temp_time_end = chrono::high_resolution_clock::now();
        temp_spent_time = get_time_spent("", temp_time_begin, temp_time_end);
        temp_unique += temp_spent_time;
        unique_time += temp_spent_time;
        temp_deduplication += temp_spent_time;
        output.deduplication_time += temp_spent_time;
        time_point_begin = chrono::high_resolution_clock::now();
        hipFree(result);
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_memory_clear += spent_time;
        output.memory_clear_time += spent_time;
        time_point_begin = chrono::high_resolution_clock::now();
        checkCuda(hipMalloc((void **) &result, deduplicated_result_rows * sizeof(Entity)));
        // Copy the deduplicated concatenated result to result
        thrust::copy(thrust::device, concatenated_result,
                     concatenated_result + deduplicated_result_rows, result);
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_union += spent_time;
        output.union_time += spent_time; // changed this time from deduplication to union
        t_delta_rows = projection_rows;
        time_point_begin = chrono::high_resolution_clock::now();
        // Clear intermediate memory
        hipFree(join_result);
        hipFree(offset);
        hipFree(concatenated_result);
        time_point_end = chrono::high_resolution_clock::now();
        spent_time = get_time_spent("", time_point_begin, time_point_end);
        temp_memory_clear += spent_time;
        output.memory_clear_time += spent_time;

        if (result_rows == deduplicated_result_rows) {
            iterations++;
            break;
        }
        result_rows = deduplicated_result_rows;
        iterations++;
    }
    time_point_begin = chrono::high_resolution_clock::now();
    checkCuda(hipHostMalloc((void **) &result_host, result_rows * sizeof(Entity)));
    hipMemcpy(result_host, result, result_rows * sizeof(Entity),
               hipMemcpyDeviceToHost);
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.union_time += spent_time;
    time_point_begin = chrono::high_resolution_clock::now();
    // Clear memory
    hipFree(t_delta);
    hipFree(result);
    hipFree(hash_table);
    hipHostFree(result_host);
    time_point_end = chrono::high_resolution_clock::now();
    spent_time = get_time_spent("", time_point_begin, time_point_end);
    output.memory_clear_time += spent_time;
    double calculated_time = output.initialization_time +
                             output.read_time + output.reverse_time + output.hashtable_build_time + output.join_time +
                             output.projection_time +
                             output.union_time + output.deduplication_time + output.memory_clear_time;
    cout << endl;
    cout << "| Dataset | Number of rows | TC size | Iterations | Blocks x Threads | Time (s) |" << endl;
    cout << "| --- | --- | --- | --- | --- | --- |" << endl;
    cout << "| " << dataset_name << " | " << relation_rows << " | " << result_rows;
    cout << fixed << " | " << iterations << " | ";
    cout << fixed << grid_size << " x " << block_size << " | " << calculated_time << " |\n" << endl;
    output.block_size = block_size;
    output.grid_size = grid_size;
    output.input_rows = relation_rows;
    output.load_factor = load_factor;
    output.hashtable_rows = hash_table_rows;
    output.dataset_name = dataset_name;
    output.total_time = calculated_time;

    cout << endl;
    cout << "Initialization: " << output.initialization_time;
    cout << ", Read: " << output.read_time << endl;
    cout << "Hashtable rate: " << output.hashtable_build_rate << " keys/s, time: ";
    cout << output.hashtable_build_time << endl;
    cout << "Join: " << output.join_time << endl;
    cout << "Deduplication: " << output.deduplication_time;
    cout << " (sort: " << sort_time << ", unique: " << unique_time << ")" << endl;
    cout << "Memory clear: " << output.memory_clear_time << endl;
    cout << "Union: " << output.union_time << " (merge: " << merge_time << ")" << endl;
    cout << "Total: " << output.total_time << endl;
}

void run_benchmark(int grid_size, int block_size, double load_factor) {
    // Variables to store device information
    int device_id;
    int number_of_sm;

    // Get the current CUDA device
    hipGetDevice(&device_id);
    // Get the number of streaming multiprocessors (SM) on the device
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount, device_id);

    // Set locale for printing numbers with commas as thousands separator
    std::locale loc("");
    std::cout.imbue(loc);
    std::cout << std::fixed;
    std::cout << std::setprecision(4);

    // Separator character for dataset names and paths
    char separator = '\t';

    // Array of dataset names and paths, filename pattern: data_<number_of_rows>.txt
    string datasets[] = {
            "OL.cedge_initial", "data_7035.txt"
    };

    // Iterate over the datasets array
    // Each iteration processes a dataset
    for (int i = 0; i < sizeof(datasets) / sizeof(datasets[0]); i += 2) {
        const char *data_path, *dataset_name;
        // Extract the dataset name and path from the array
        dataset_name = datasets[i].c_str();
        data_path = datasets[i + 1].c_str();

        // Get the row size of the dataset
        long int row_size = get_row_size(data_path);

        // Print benchmark information for the current dataset
        cout << "Benchmark for " << dataset_name << endl;
        cout << "----------------------------------------------------------" << endl;

        // Run the GPU graph processing function with the dataset parameters
        gpu_tc(data_path, separator,
               row_size, load_factor,
               grid_size, block_size, dataset_name, number_of_sm);

        cout << endl;
    }
}


int main() {
    run_benchmark(0, 0, 0.4);
    return 0;
}

/*
Run instructions:
make run
*/
